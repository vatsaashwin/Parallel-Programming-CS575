#include "hip/hip_runtime.h"
// Array multiplication: C = A * B:

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE
#define BLOCKSIZE		16		// number of threads per block
#endif

#ifndef SIZE
#define SIZE			16*1024	// array size
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

// ranges for the random numbers:
const float XCMIN =	 0.0;
const float XCMAX =	 2.0;
const float YCMIN =	 0.0;
const float YCMAX =	 2.0;
const float RMIN  =	 0.5;
const float RMAX  =	 2.0;

// monte carlo (CUDA Kernel) on the device
__global__  void MonteCarlo( float *xcs, float *ycs, float *rs, int *numHits )
{
    __shared__ int prods[BLOCKSIZE];

	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

    float xc = xcs[gid];
    float yc = ycs[gid];
    float  r =  rs[gid];

    // solve for the intersection using the quadratic formula:
    float a = 2.;
    float b = -2.*( xc + yc );
    float c = xc*xc + yc*yc - r*r;
    float d = b*b - 4.*a*c;

    if( d < 0. )
    {
        prods[tnum] = 0;
    }
    else
    {
        // hits the circle:
        d = sqrt( d );
        float t1 = (-b + d ) / ( 2.*a );	
        float t2 = (-b - d ) / ( 2.*a );	
        float tmin = t1 < t2 ? t1 : t2;		

        if( tmin < 0. )
        {
            prods[tnum] = 0;
        }
        else
        {

            float xcir = tmin;
            float ycir = tmin;


            float nx = xcir - xc;
            float ny = ycir - yc;
            float n = sqrt( nx*nx + ny*ny );
            nx /= n;	
            ny /= n;	


            float inx = xcir - 0.;
            float iny = ycir - 0.;
            float in = sqrt( inx*inx + iny*iny );
            inx /= in;	// unit vector
            iny /= in;	// unit vector

            // get the outgoing (bounced) vector:
            float dot = inx*nx + iny*ny;
            float outx = inx - 2.*nx*dot;	
            float outy = iny - 2.*ny*dot;	

            // find out if it hits the infinite plate:
            float t = ( 0. - ycir ) / outy;

            if( t < 0. )
            {
                prods[tnum] = 0;
            }
            else
            {
                prods[tnum] = 1;
            }
        }
    }

	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			prods[tnum] += prods[tnum + offset];
		}
	}

	__syncthreads();
	if (tnum == 0)
		numHits[wgNum] = prods[0];
}

// helper functions
float Ranf( float low, float high )
{
        float r = (float) rand();               // 0 - RAND_MAX
        float t = r  /  (float) RAND_MAX;       // 0. - 1.

        return   low  +  t * ( high - low );
}

int Ranf( int ilow, int ihigh )
{
        float low = (float)ilow;
        float high = ceil( (float)ihigh );

        return (int) Ranf(low,high);
}

// main program:

int
main( int argc, char* argv[ ] )
{
	int dev = findCudaDevice(argc, (const char **)argv);

	// allocate host memory:
    
	float * hxcs = new float [ SIZE ];
	float * hycs = new float [ SIZE ];
	float * hrs = new float [ SIZE ];
    int * hnumHits = new int [ SIZE/BLOCKSIZE ];
    
    // fill the random-value arrays:
    for( int n = 0; n < SIZE; n++ )
    {
        hxcs[n] = Ranf( XCMIN, XCMAX );
        hycs[n] = Ranf( YCMIN, YCMAX );
        hrs[n] = Ranf(  RMIN,  RMAX );
    }
    
	// allocate device memory:

    float *dxcs, *dycs, *drs;
    int *dnumHits;
    
	dim3 dimsxcs( SIZE, 1, 1 );
	dim3 dimsycs( SIZE, 1, 1 );
	dim3 dimsrs( SIZE, 1, 1 );
    dim3 dimsnumHits( SIZE, 1, 1);
    

	hipError_t status;

	status = hipMalloc( reinterpret_cast<void **>(&dxcs), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dycs), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&drs), SIZE*sizeof(float) );
		checkCudaErrors( status );
    status = hipMalloc( reinterpret_cast<void **>(&dnumHits), (SIZE/BLOCKSIZE)*sizeof(int) );
		checkCudaErrors( status );


	// copy host memory to the device:

	status = hipMemcpy( dxcs, hxcs, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dycs, hycs, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
    status = hipMemcpy( drs, hrs, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( SIZE / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:

    MonteCarlo<<< grid, threads >>>( dxcs, dycs, drs, dnumHits );

	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double trialsPerSecond = (float)SIZE / secondsTotal;
	double megaTrialsPerSecond = trialsPerSecond / 1000000.;
	fprintf( stderr, "SIZE = %10d, MegaTrials/Second = %10.2lf\n", SIZE, megaTrialsPerSecond );

	// copy result from the device to the host:

	status = hipMemcpy( hnumHits, dnumHits, (SIZE/BLOCKSIZE)*sizeof(int), hipMemcpyDeviceToHost );
		checkCudaErrors( status );
    
    int sum = 0.;
	for(int i = 0; i < SIZE/BLOCKSIZE; i++ )
	{
		sum += hnumHits[i];
	}
    float prob = (float)(sum) / (float)(SIZE);
    fprintf( stderr, "\nNumHit=%10d\n", sum );
	fprintf( stderr, "\nProbability=%8.4lf\n", prob );
    
	// clean up memory:
    delete [ ] hxcs;
    delete [ ] hycs;
    delete [ ] hrs;
    delete [ ] hnumHits;

    status = hipFree( dxcs );
		checkCudaErrors( status );
	status = hipFree( dycs );
		checkCudaErrors( status );
	status = hipFree( drs );
		checkCudaErrors( status );
    status = hipFree( dnumHits );
		checkCudaErrors( status );

	return 0;
}